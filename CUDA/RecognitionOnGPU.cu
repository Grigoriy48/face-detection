#include "hip/hip_runtime.h"
#include "RecognitionOnGPU.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""
#include <assert.h>
#include <stdlib.h>
#include <math.h>


void ParallelizationElements(TXMLCascade* cascade, TStage* stages, TFeature* features, TRect* rects,
	unsigned short width_original, unsigned short height_original, unsigned char* char_picture_original,
	unsigned short byte_pixel_original, unsigned short bit_width_original, float step_factor,
	int* pixel_intensity, int* pixel_intensity_sqr) {
	float factor = 1.0;
	float *dev_factor;
	hipMalloc((void**)&dev_factor, sizeof(float));
	hipMemcpy(dev_factor, &factor, sizeof(float), hipMemcpyHostToDevice);
	TXMLCascade* dev_cascade;
	hipMalloc((void**)&dev_cascade, sizeof(TXMLCascade));
	hipMemcpy(dev_cascade, cascade, sizeof(TXMLCascade), hipMemcpyHostToDevice);
	TStage* dev_stages;
	hipMalloc((void**)&dev_stages, sizeof(TStage) * 30);
	hipMemcpy(dev_stages, stages, sizeof(TStage) * 30, hipMemcpyHostToDevice);
	TFeature* dev_features;
	hipMalloc((void**)&dev_features, sizeof(TFeature) * 7000);
	hipMemcpy(dev_features, features, sizeof(TFeature) * 7000, hipMemcpyHostToDevice);
	TRect* dev_rects;
	hipMalloc((void**)&dev_rects, sizeof(TRect) * 11000);
	hipMemcpy(dev_rects, rects, sizeof(TRect) * 11000, hipMemcpyHostToDevice);
	unsigned short* dev_width_original;
	hipMalloc((void**)&dev_width_original, sizeof(unsigned short));
	hipMemcpy(dev_width_original, &width_original, sizeof(unsigned short), hipMemcpyHostToDevice);
	unsigned short* dev_height_original;
	hipMalloc((void**)&dev_height_original, sizeof(unsigned short));
	hipMemcpy(dev_height_original, &height_original, sizeof(unsigned short), hipMemcpyHostToDevice);
	unsigned char* dev_char_picture_original;
	hipMalloc((void**)&dev_char_picture_original, sizeof(unsigned char) * 3 * height_original*width_original);
	hipMemcpy(dev_char_picture_original, char_picture_original, sizeof(unsigned char) * 3 * height_original*width_original, hipMemcpyHostToDevice);
	unsigned short* dev_byte_pixel_original;
	hipMalloc((void**)&dev_byte_pixel_original, sizeof(unsigned short));
	hipMemcpy(dev_byte_pixel_original, &byte_pixel_original, sizeof(unsigned short), hipMemcpyHostToDevice);
	unsigned short* dev_bit_width_original;
	hipMalloc((void**)&dev_bit_width_original, sizeof(unsigned short));
	hipMemcpy(dev_bit_width_original, &bit_width_original, sizeof(unsigned short), hipMemcpyHostToDevice);
	float* dev_step_factor;
	hipMalloc((void**)&dev_step_factor, sizeof(float));
	hipMemcpy(dev_step_factor, &step_factor, sizeof(float), hipMemcpyHostToDevice);
	int* dev_pixel_intensity;
	hipMalloc((void**)&dev_pixel_intensity, width_original*height_original * 4);
	hipMemcpy(dev_pixel_intensity, pixel_intensity, width_original*height_original * 4, hipMemcpyHostToDevice);
	int* dev_pixel_intensity_sqr;
	hipMalloc((void**)&dev_pixel_intensity_sqr, width_original*height_original * 4);
	hipMemcpy(dev_pixel_intensity_sqr, pixel_intensity_sqr, width_original*height_original * 4, hipMemcpyHostToDevice);
	int window_w = cascade->window_w_mini;
	int window_h = cascade->window_h_mini;
	do {
		ParallelizationElementsKernel <<< 512, 512 >>>(dev_cascade, dev_stages, dev_features, dev_rects,
			dev_width_original, dev_height_original, dev_char_picture_original, dev_byte_pixel_original,
			dev_bit_width_original, dev_step_factor, dev_pixel_intensity, dev_pixel_intensity_sqr,
			dev_factor);
		factor *= step_factor;
		hipMemcpy(dev_factor, &factor, sizeof(float), hipMemcpyHostToDevice);
		window_w = floor(cascade->window_w_mini*factor);
		window_h = floor(cascade->window_h_mini*factor);
	} while (min(width_original, height_original) >= min(window_w, window_h));

	printf("Error: %s\n", hipGetErrorString(hipGetLastError()));
	
	hipMemcpy(char_picture_original, dev_char_picture_original, sizeof(unsigned char) * 3 * height_original*width_original, hipMemcpyDeviceToHost);
}

__global__ void ParallelizationElementsKernel(TXMLCascade* cascade, TStage* stages, TFeature* features, TRect* rects,
	unsigned short* width_original, unsigned short* height_original, unsigned char* char_picture_original,
	unsigned short* byte_pixel_original, unsigned short* bit_width_original, float* step_factor,
	int* pixel_intensity, int* pixel_intensity_sqr, float* factor) {
	int window_w = floor(cascade->window_w_mini**factor);
	int window_h = floor(cascade->window_h_mini**factor);
	unsigned short x1, y1;
	int x_step = ((1) >= (((4)<((window_w) / 10) ? (4) : ((window_w) / 10))) ? (1) : (((4)<((window_w) / 10) ? (4) : ((window_w) / 10))));
	int y_step = ((1) >= (((4)<((window_h) / 10) ? (4) : ((window_h) / 10))) ? (1) : (((4)<((window_h) / 10) ? (4) : ((window_h) / 10))));
	y1 = y_step *  blockIdx.x;
	if (y1 < *height_original - window_h) {
		int	x_thread_step = 0;
		for (x1 = 0; x1 < *width_original - window_w; x_thread_step += 512) {
			x1 = x_thread_step + x_step *  threadIdx.x;
			if (x1 < 512 + x_thread_step) {
				float mean = intensity_window_device(x1, y1, window_w, window_h, pixel_intensity, *width_original) * 1 / float(window_w*window_h);
				float variance = sqr_intensity_window_device(x1, y1, window_w, window_h, pixel_intensity_sqr, *width_original) * 1 / float(window_w*window_h) - (mean*mean);
				float stddev = 1.0;
				stddev = sqrt(variance);
				if (stddev > 10.0) {
					bool f_failed = false;
					for (int i_stage = 0; i_stage < cascade->n_stages; i_stage = i_stage + 1) {
						float sum_stage = 0.0;
						for (int i_feature = stages[i_stage].i_feature_start;
							i_feature <= stages[i_stage].i_feature_finish;
							i_feature = i_feature + 1) {
							int sum_feature = 0.0;
							for (int i_rect = features[i_feature].i_rect_start;
								i_rect <= features[i_feature].i_rect_finish;
								i_rect = i_rect + 1) {
								sum_feature += (intensity_window_device(x1 + rects[i_rect].x**factor, y1 + rects[i_rect].y**factor,
									rects[i_rect].w**factor, rects[i_rect].h**factor, pixel_intensity, *width_original)*rects[i_rect].weight);
							}
							if (sum_feature * 1 / float(window_w*window_h) < features[i_feature].feature_threshold * stddev)
								sum_stage += features[i_feature].left_val;
							else
								sum_stage += features[i_feature].right_val;
						}
						if (sum_stage < stages[i_stage].stage_threshold) {
							f_failed = true;
							break;
						}
					}
					if (f_failed == false) {
						printf("%d %d %d %d  \n", x1, y1, x1 + window_w, y1 + window_h);
						unsigned short x2 = x1 + window_w;
						unsigned short y2 = y1 + window_h;
						for (int x = x1; x <= x2; x++) {
							char_picture_original[*bit_width_original*(*height_original - y1) + *byte_pixel_original*x + 0] = 0x00;
							char_picture_original[*bit_width_original*(*height_original - y1) + *byte_pixel_original*x + 1] = 0x00;
							char_picture_original[*bit_width_original*(*height_original - y1) + *byte_pixel_original*x + 2] = 0x00;
						}
						for (int x = x1; x <= x2; x++) {
							char_picture_original[*bit_width_original*(*height_original - y2) + *byte_pixel_original*x + 0] = 0x00;
							char_picture_original[*bit_width_original*(*height_original - y2) + *byte_pixel_original*x + 1] = 0x00;
							char_picture_original[*bit_width_original*(*height_original - y2) + *byte_pixel_original*x + 2] = 0x00;
						}
						for (int y = y1; y <= y2; y++) {
							char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x1 + 0] = 0x00;
							char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x1 + 1] = 0x00;
							char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x1 + 2] = 0x00;
						}
						for (int y = y1; y <= y2; y++) {
							char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x2 + 0] = 0x00;
							char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x2 + 1] = 0x00;
							char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x2 + 2] = 0x00;
						}
					}
				}
			}
		}
	}
}

void ParallelizationScale(TXMLCascade* cascade, TStage* stages, TFeature* features, TRect* rects,
	unsigned short width_original, unsigned short height_original, unsigned char* char_picture_original,
	unsigned short byte_pixel_original, unsigned short bit_width_original, float step_factor,
	int* pixel_intensity, int* pixel_intensity_sqr, int* mas_pointer, int** mas) {

	unsigned char* dev_char_picture_original;
	hipMalloc((void**)&dev_char_picture_original, sizeof(unsigned char) * 3 * height_original*width_original);
	hipMemcpy(dev_char_picture_original, char_picture_original, sizeof(unsigned char) * 3 * height_original*width_original, hipMemcpyHostToDevice);

	int* dev_mas_pointer;
	hipMalloc((void**)&dev_mas_pointer, sizeof(int));

	int* dev_mas;
	hipMalloc((void**)&dev_mas, sizeof(int) * 512);

	float factor = 1.0;
	float *dev_factor;
	hipMalloc((void**)&dev_factor, sizeof(float));
	hipMemcpy(dev_factor, &factor, sizeof(float), hipMemcpyHostToDevice);
	TXMLCascade* dev_cascade;
	hipMalloc((void**)&dev_cascade, sizeof(TXMLCascade));
	hipMemcpy(dev_cascade, cascade, sizeof(TXMLCascade), hipMemcpyHostToDevice);
	TStage* dev_stages;
	hipMalloc((void**)&dev_stages, sizeof(TStage) * 30);
	hipMemcpy(dev_stages, stages, sizeof(TStage) * 30, hipMemcpyHostToDevice);
	TFeature* dev_features;
	hipMalloc((void**)&dev_features, sizeof(TFeature) * 7000);
	hipMemcpy(dev_features, features, sizeof(TFeature) * 7000, hipMemcpyHostToDevice);
	TRect* dev_rects;
	hipMalloc((void**)&dev_rects, sizeof(TRect) * 11000);
	hipMemcpy(dev_rects, rects, sizeof(TRect) * 11000, hipMemcpyHostToDevice);
	unsigned short* dev_width_original;
	hipMalloc((void**)&dev_width_original, sizeof(unsigned short));
	hipMemcpy(dev_width_original, &width_original, sizeof(unsigned short), hipMemcpyHostToDevice);
	unsigned short* dev_height_original;
	hipMalloc((void**)&dev_height_original, sizeof(unsigned short));
	hipMemcpy(dev_height_original, &height_original, sizeof(unsigned short), hipMemcpyHostToDevice);
	unsigned short* dev_byte_pixel_original;
	hipMalloc((void**)&dev_byte_pixel_original, sizeof(unsigned short));
	hipMemcpy(dev_byte_pixel_original, &byte_pixel_original, sizeof(unsigned short), hipMemcpyHostToDevice);
	unsigned short* dev_bit_width_original;
	hipMalloc((void**)&dev_bit_width_original, sizeof(unsigned short));
	hipMemcpy(dev_bit_width_original, &bit_width_original, sizeof(unsigned short), hipMemcpyHostToDevice);
	float* dev_step_factor;
	hipMalloc((void**)&dev_step_factor, sizeof(float));
	hipMemcpy(dev_step_factor, &step_factor, sizeof(float), hipMemcpyHostToDevice);
	int* dev_pixel_intensity;
	hipMalloc((void**)&dev_pixel_intensity, width_original*height_original * 4);
	hipMemcpy(dev_pixel_intensity, pixel_intensity, width_original*height_original * 4, hipMemcpyHostToDevice);
	int* dev_pixel_intensity_sqr;
	hipMalloc((void**)&dev_pixel_intensity_sqr, width_original*height_original * 4);
	hipMemcpy(dev_pixel_intensity_sqr, pixel_intensity_sqr, width_original*height_original * 4, hipMemcpyHostToDevice);
	int window_w, window_h;
	float scale[100];
	int k = 0;
	do {
		window_w = (int)floor(cascade->window_w_mini*factor);
		window_h = (int)floor(cascade->window_h_mini*factor);
		scale[k] = factor;
		factor *= step_factor;
		k++;
	} while (min(width_original, height_original) >= min(window_w, window_h));
	printf("block = %d;  \n", k);
	float* dev_scale;
	hipMalloc((void**)&dev_scale, sizeof(float) * 100);
	hipMemcpy(dev_scale, scale, sizeof(float) * 100, hipMemcpyHostToDevice);


	ParallelizationScaleKernel <<< k, 512 >>>(dev_cascade, dev_stages, dev_features, dev_rects,
		dev_width_original, dev_height_original, dev_char_picture_original, dev_byte_pixel_original,
		dev_bit_width_original, dev_step_factor, dev_pixel_intensity, dev_pixel_intensity_sqr,
		dev_factor, dev_scale, dev_mas_pointer, dev_mas);
	

	hipDeviceSynchronize();

	printf("Error 1: %s\n", hipGetErrorString(hipGetLastError()));
	
	hipMemcpy(char_picture_original, dev_char_picture_original, sizeof(unsigned char) * 3 * height_original*width_original, hipMemcpyDeviceToHost);

		/*printf("Error 2: %s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy( &mas, dev_mas, sizeof(int)*512*k,hipMemcpyDeviceToHost);

	printf("Error 3: %s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy(mas_pointer, dev_mas_pointer, sizeof(int), hipMemcpyDeviceToHost);

	printf("Error 4: %s\n", hipGetErrorString(hipGetLastError()));*/

}

__global__ void ParallelizationScaleKernel(TXMLCascade* cascade, TStage* stages, TFeature* features, TRect* rects,
	unsigned short* width_original, unsigned short* height_original, unsigned char* char_picture_original,
	unsigned short* byte_pixel_original, unsigned short* bit_width_original, float* step_factor,
	int* pixel_intensity, int* pixel_intensity_sqr, float* factor, float* scale, int* mas_pointer, int* mas) {
	int window_w = floor(cascade->window_w_mini*scale[blockIdx.x]);
	int window_h = floor(cascade->window_h_mini*scale[blockIdx.x]);
	unsigned short x1, y1;
	int x_step = ((1) >= (((4)<((window_w) / 10) ? (4) : ((window_w) / 10))) ? (1) : (((4)<((window_w) / 10) ? (4) : ((window_w) / 10))));
	int y_step = ((1) >= (((4)<((window_h) / 10) ? (4) : ((window_h) / 10))) ? (1) : (((4)<((window_h) / 10) ? (4) : ((window_h) / 10))));
	for (y1 = 0; y1 <= *height_original - 1 - window_h; y1 += y_step) {
		x1 = x_step *  threadIdx.x;
		if (x1 < *width_original - 1 - window_w) {
			float mean = intensity_window_device(x1, y1, window_w, window_h, pixel_intensity, *width_original) * 1 / float(window_w*window_h);
			float variance = sqr_intensity_window_device(x1, y1, window_w, window_h, pixel_intensity_sqr, *width_original) * 1 / float(window_w*window_h) - (mean*mean);
			float stddev = 1.0;
			stddev = sqrt(variance);
			if (stddev < 10.0)
				continue;
			int f_failed = 0;
			for (int i_stage = 0; i_stage < cascade->n_stages; i_stage = i_stage + 1) {
				float sum_stage = 0.0;
				for (int i_feature = stages[i_stage].i_feature_start;
					i_feature <= stages[i_stage].i_feature_finish;
					i_feature = i_feature + 1) {
					int sum_feature = 0.0;
					for (int i_rect = features[i_feature].i_rect_start;
						i_rect <= features[i_feature].i_rect_finish;
						i_rect = i_rect + 1) {
						sum_feature += (intensity_window_device(x1 + rects[i_rect].x*scale[blockIdx.x], y1 + rects[i_rect].y*scale[blockIdx.x],
							rects[i_rect].w*scale[blockIdx.x], rects[i_rect].h*scale[blockIdx.x], pixel_intensity,
							*width_original)*rects[i_rect].weight);
					}
					float leafth = features[i_feature].feature_threshold * stddev;
					if (sum_feature * 1 / float(window_w*window_h) < leafth)
						sum_stage += features[i_feature].left_val;
					else
						sum_stage += features[i_feature].right_val;
				}
				if (sum_stage < stages[i_stage].stage_threshold) {
					f_failed = 1;
					break;
				}
			}
			if (f_failed == false) {
				printf("%d %d %d %d  \n", x1, y1, x1 + window_w, y1 + window_h);
				unsigned short x2 = x1 + window_w;
				unsigned short y2 = y1 + window_h;

				/*		atomicAdd( &mas[*mas_pointer], x1);
				atomicAdd( &mas[*mas_pointer+1] , y1 );
				atomicAdd( &mas[*mas_pointer+2] , x2 );
				atomicAdd( &mas[*mas_pointer+3] , y2 );

				atomicAdd(mas_pointer, 4);
				printf("dev_mas_pointer=%d \n", *mas_pointer);*/


				for (int x = x1; x <= x2; x++) {
					char_picture_original[*bit_width_original*(*height_original - y1) + *byte_pixel_original*x + 0] = 0x00;
					char_picture_original[*bit_width_original*(*height_original - y1) + *byte_pixel_original*x + 1] = 0x00;
					char_picture_original[*bit_width_original*(*height_original - y1) + *byte_pixel_original*x + 2] = 0x00;
				}
				for (int x = x1; x <= x2; x++) {
					char_picture_original[*bit_width_original*(*height_original - y2) + *byte_pixel_original*x + 0] = 0x00;
					char_picture_original[*bit_width_original*(*height_original - y2) + *byte_pixel_original*x + 1] = 0x00;
					char_picture_original[*bit_width_original*(*height_original - y2) + *byte_pixel_original*x + 2] = 0x00;
				}
				for (int y = y1; y <= y2; y++) {
					char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x1 + 0] = 0x00;
					char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x1 + 1] = 0x00;
					char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x1 + 2] = 0x00;
				}
				for (int y = y1; y <= y2; y++) {
					char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x2 + 0] = 0x00;
					char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x2 + 1] = 0x00;
					char_picture_original[*bit_width_original*(*height_original - y) + *byte_pixel_original*x2 + 2] = 0x00;
				}
			} 
		}
	}
}

__device__ int intensity_window_device(int x, int y, int w_window, int h_window, int* pixel_intensity, unsigned short width_original) {
	int pixel_4 = pixel_intensity[width_original*(y + h_window) + (x + w_window)];
	int pixel_1 = pixel_intensity[width_original*(y + 0) + (x + 0)];
	int pixel_2 = pixel_intensity[width_original*(y + 0) + (x + w_window)];
	int pixel_3 = pixel_intensity[width_original*(y + h_window) + (x + 0)];
	return (pixel_4 + pixel_1 - pixel_2 - pixel_3);
}

__device__ int sqr_intensity_window_device(int x, int y, int w_window, int h_window, int* pixel_intensity_sqr, unsigned short width_original) {
	int pixel_4 = pixel_intensity_sqr[width_original*(y + h_window) + (x + w_window)];
	int pixel_1 = pixel_intensity_sqr[width_original*(y + 0) + (x + 0)];
	int pixel_2 = pixel_intensity_sqr[width_original*(y + 0) + (x + w_window)];
	int pixel_3 = pixel_intensity_sqr[width_original*(y + h_window) + (x + 0)];
	return (pixel_4 + pixel_1 - pixel_2 - pixel_3);
}
