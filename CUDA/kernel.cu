#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

#ifndef __OPENCV_OLD_HIGHGUI_H__
#define __OPENCV_OLD_HIGHGUI_H__

#endif
#include "TreatmentOnCPU.h"

int main()
{
	printf("=============================================================\n");
	printf("===================== FACE DETECTION ========================\n");
	printf("=============================================================\n\n\n");

	TreatmentOnCPU *TOC = new TreatmentOnCPU();
	TOC->Treatment();
}
