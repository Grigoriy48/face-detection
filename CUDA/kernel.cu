#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

#ifndef __OPENCV_OLD_HIGHGUI_H__
#define __OPENCV_OLD_HIGHGUI_H__

#endif
#include "TreatmentOnCPU.h"
#include "SkinColorDetection.h"

int main()
{
	printf("=============================================================\n");
	printf("===================== FACE DETECTION ========================\n");
	printf("=============================================================\n\n\n");

	

	printf("=============================================================\n\n\n");
	printf("skin color detection method:\n");
	printf("1 - RGB\n");
	printf("2 - Log Opponent & YIQ\n");
	printf("3 - HSV\n");
	printf("4 - TSL\n");
	printf("5 - all methods\n");

	int method;
	scanf("%d", &method);
	SkinColorDetection skinColorDetection(method);

	
	//TreatmentOnCPU *TOC = new TreatmentOnCPU();
	//TOC->Treatment();
}
